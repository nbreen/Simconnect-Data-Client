#include "hip/hip_runtime.h"
#include <string>
#include <malloc.h>
#include <iostream>
#include "../simconnectData.pb.h"

#define N 26

__global__ void cudaProcess(double *toProcess) {
  toProcess[blockIdx.x] = toProcess[blockIdx.x];
}

double* toArray(simConnect::simData toConvert) {
  double *converted = (double *) malloc(sizeof(double) * N);
  converted[0] = toConvert.dabsolutetime();
  converted[1] = toConvert.dtime();
  converted[2] = toConvert.udayoftheyear();
  converted[3] = toConvert.uyear();
  converted[4] = toConvert.umonthoftheyear();
  converted[5] = toConvert.udayofthemonth();
  converted[6] = toConvert.udayoftheweek();
  converted[7] = toConvert.utimezoneoffset();
  converted[8] = toConvert.dsimtime();
  converted[9] = toConvert.dlatitude();
  converted[10] = toConvert.dlongitude();
  converted[11] = toConvert.daltitude();
  converted[12] = toConvert.dpitch();
  converted[13] = toConvert.dbank();
  converted[14] = toConvert.dheading();
  converted[15] = toConvert.dvelocityx();
  converted[16] = toConvert.dvelocityy();
  converted[17] = toConvert.dvelocityz();
  converted[18] = toConvert.dtemperature();
  converted[19] = toConvert.dairpressure();
  converted[20] = toConvert.dairdensity();
  converted[21] = toConvert.dwindvelocity();
  converted[22] = toConvert.dwinddirection();
  converted[23] = toConvert.dwindx();
  converted[24] = toConvert.dwindy();
  converted[25] = toConvert.dwindz();

  return converted;
}

int main(void) {
  simConnect::simData toProcess;
  std::string toStr("This is a test");
  double *convertedData;
  double *d_convertedData;
  double *processedData;
  int size = N * sizeof(double);

  toProcess.set_sztitle(toStr);
  toProcess.set_dabsolutetime(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dtime(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_udayoftheyear(5);
  toProcess.set_uyear(5);
  toProcess.set_umonthoftheyear(5);
  toProcess.set_udayofthemonth(5);
  toProcess.set_udayoftheweek(5);
  toProcess.set_utimezoneoffset(5);
  toProcess.set_dsimtime(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dlatitude(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dlongitude(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_daltitude(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dpitch(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dbank(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dheading(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dvelocityx(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dvelocityy(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dvelocityz(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dtemperature(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dairpressure(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dairdensity(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dwindvelocity(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dwinddirection(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dwindx(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dwindy(16.44444444444444444444444444444444444444444444444444444444444);
  toProcess.set_dwindz(16.44444444444444444444444444444444444444444444444444444444444);

  /* In order to process the data we first need it in an array for easier access */
  convertedData = toArray(toProcess);
  /* Make space for our processed data */
  processedData = (double *)malloc(size);
  /* Make space on GPU memory for our array */
  hipMalloc((void **)&d_convertedData, size);
  /* Copy our array from host to device */
  hipMemcpy(d_convertedData, convertedData, size, hipMemcpyHostToDevice);
  /* Launch kernel on the GPU to process the data we just copied */
  cudaProcess<<<N,1>>>(d_convertedData);
  /* Copy our pocessed data back from device to host */
  hipMemcpy(processedData, d_convertedData, size, hipMemcpyDeviceToHost);
  /* Make sure any device jobs finish before we cleanup and exit */
  hipDeviceSynchronize();

  std::cout << "Object sztitle: " << toStr <<std::endl;
  for (int i = 0; i < N; i++ ) {
    printf("processedData %d: %f \n", i, processedData[i]);
  }

  free(convertedData);
  free(processedData);
  hipFree(d_convertedData);

  return 0;
}
