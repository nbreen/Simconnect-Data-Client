#include "hip/hip_runtime.h"
#include "cudaProcess.h"

void logToFile(simConnect::simData payload, double *convertedData, processedData_t *processedData ) {
  FILE *output = fopen("out.txt", "a+");
  std::string  printString ("");
  
  fprintf(output, "Object sztitle: %s\n", payload.sztitle().c_str());
  
  for (int i = 0; i < N; i++ ) {
    switch(i) {
      case dAbsoluteTime:
        // processedData->absTime->year
        // processedData->absTime->month
        // processedData->absTime->day
        // processedData->absTime->hour
        // processedData->absTime->min
        printString = "Absolute time Unprocessed: " + 
                      std::to_string(convertedData[dAbsoluteTime]);
        printString.append( " Processed: ");
        printString += std::to_string(processedData->absTime->sec);
        printString.append(" Seconds");
        break;
      case dTime:
        printString = "Zulu Time Unprocessed: " +
                      std::to_string(convertedData[dTime]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->zulu->hour);
        printString.append(":");
        printString += std::to_string(processedData->zulu->min); 
        printString.append(":");
        printString += std::to_string(processedData->zulu->sec);
        break;
      case uSimOnGround: {
        std::string onG;

        if (processedData->onGround) {
          onG = "True";
        } else {
          onG = "False";
        }
        printString = "Sim On Ground Unprocessed: " + 
                      std::to_string(convertedData[uSimOnGround]);
        printString.append(" Processed: ");
        printString += onG;
        break;
      }
      case dAltitude:
        printString = "Altitude Unprocessed: " +
                      std::to_string(convertedData[dAltitude]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->altitude);
        printString.append(" Feet");
        break;
      case dHeading:
        printString = "Heading Unprocessed: " +
                      std::to_string(convertedData[dHeading]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->heading); 
        printString.append(" Degrees");
        break;
      case dSpeed:
        printString = "Speed  Unprocessed: " +
                      std::to_string(convertedData[dSpeed]);
        printString.append(" Processed: ");
        printString += std::to_string(processedData->speed); 
        printString.append(" Knots");
        break;
      case dVerticalSpeed:
        printString = "Vertical Speed Unprocessed: " +
                      std::to_string(convertedData[dVerticalSpeed]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->verticalSpeed); 
        printString.append(" Feet Per Minute");
        break;
      case dGpsEta:
        printString = "GPS ETA  Unprocessed: " +
                      std::to_string(convertedData[dGpsEta]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->gpsEta->hour);
        printString.append(":"); 
        printString += std::to_string(processedData->gpsEta->min);
        printString.append(":");
        printString +=std::to_string(processedData->gpsEta->sec);
        break;
      case dLatitude:
        printString = "Latitude Unprocessed: " +
                      std::to_string(convertedData[dLatitude]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->lat);
        printString.append(" Degrees");
        break;
      case dLongitude:
        printString = "Longitude Unprocessed: " +
                      std::to_string(convertedData[dLongitude]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->longi);
        printString.append(" Degrees");
        break;
      case dSimTime:
        printString = "Sim time  Unprocessed: " +
                      std::to_string(convertedData[dSimTime]); 
        printString.append(" Processed: ");
        printString += std::to_string(processedData->simulationTime->hour);
        printString.append(":"); 
        printString += std::to_string(processedData->simulationTime->min);
        printString.append(":");
        printString += std::to_string(processedData->simulationTime->sec);
        break;
      case dTemperature:
        printString = "Temperature Unprocessed: " +
                      std::to_string(convertedData[dTemperature]);
        printString.append(" Processed: ");
        printString += std::to_string(processedData->temp);
        printString.append(" Degrees Celsius");
        break;
      case dPressure:
        printString = "Air Pressure Unprocessed: " +
                      std::to_string(convertedData[dPressure]);
        printString.append(" Processed: ");
        printString += std::to_string(processedData->pressure);
        printString.append(" In Hg");
        break;
      case dWindVelocity:
        printString = "Wind Velocity Unprocessed: " +
                      std::to_string(convertedData[dWindVelocity]);
        printString.append(" Processed: ");
        printString += std::to_string(processedData->windVelo);
        printString.append(" Knots");
        break;
      case dWindDirection:
        printString = "Wind Direction Unprocessed: " +
                      std::to_string(convertedData[dWindDirection]);
        printString.append(" Processed: ");
        printString += std::to_string(processedData->windDir);
        printString.append(" Degrees");
        break;
    }

    fprintf(output, "%s\n", printString.c_str());
  }

  fprintf(output, "\n\n");
  fclose(output);
}

double *toArray(simConnect::simData toConvert) {
  double *convertedData = (double *)malloc(sizeof(double) * N);

  for (int i = 0; i < N; i++ ) {
    switch(i) {
      case dAbsoluteTime:
        convertedData[i] = toConvert.dabsolutetime();
        break;
      case dTime:
        convertedData[i] = toConvert.dtime();
        break;
      case uSimOnGround:
        convertedData[i] = toConvert.usimonground();
        break;
      case dAltitude:
        convertedData[i] = toConvert.daltitude();
        break;
      case dHeading:
        convertedData[i] = toConvert.dheading();
        break;
      case dSpeed:
        convertedData[i] = toConvert.dspeed();
        break;
      case dVerticalSpeed:
        convertedData[i] = toConvert.dverticalspeed();
        break;
      case dGpsEta:
        convertedData[i] = toConvert.dgpseta();
        break;
      case dLatitude:
        convertedData[i] = toConvert.dlatitude();
        break;
      case dLongitude:
        convertedData[i] = toConvert.dlongitude();
        break;
      case dSimTime:
        convertedData[i] = toConvert.dsimtime();
        break;
      case dTemperature:
        convertedData[i] = toConvert.dtemperature();
        break;
      case dPressure:
        convertedData[i] = toConvert.dairpressure();
        break;
      case dWindVelocity:
        convertedData[i] = toConvert.dwindvelocity();
        break;
      case dWindDirection:
        convertedData[i] = toConvert.dwinddirection();
        break;
    }
  }

  return convertedData;
}

__global__ void cudaHandle(processedData_t *processedData, double *toProcess) {
  /* Process each field differently */
  switch(blockIdx.x) {
    case dAbsoluteTime:
      /* Seconds since 12:00AM Jan. 1 0000 -> YYYY-MM-DD-HH-MM-SS */
      //processedData->absTime->year
      //processedData->absTime->month
      //processedData->absTime->day
      //processedData->absTime->hour
      //processedData->absTime->min
      processedData->absTime->sec = toProcess[dAbsoluteTime];
      
      break;
    case dTime:
      /* Seconds -> HH-MM-SS */
      processedData->zulu->hour = toProcess[dTime] / 3600;
      processedData->zulu->min = (toProcess[dTime] / 60) - (processedData->zulu->hour * 60) ;
      processedData->zulu->sec = toProcess[dTime] - 60 - (processedData->zulu->hour * 3600) - (processedData->zulu->min * 60) ;
      
      break;
    case uSimOnGround: {
      /* int -> bool */
      int onground = __double2int_rn(toProcess[uSimOnGround]);
      
      if (onground == 1) {
        processedData->onGround = true;
      } else {
        processedData->onGround = false;
      }
      
      break;
    }
    case dAltitude:
      /* feet -> rounded feet */
      processedData->altitude = __double2int_rn(toProcess[dAltitude]);
      break;
    case dHeading:
      /* Radians -> degrees */
      processedData->heading = __double2int_rn(toProcess[dHeading] * (180/HIP_PI_F));
      break;
    case dSpeed:
      /* Knots -> rounded knots */
      processedData->speed = __double2int_rn(toProcess[dSpeed]);
      break;
    case dVerticalSpeed:
      /* Feet per Sec -> Feet Per Min */
      processedData->verticalSpeed = __double2int_rn(toProcess[dVerticalSpeed] * 60);
      break;
    case dGpsEta:
      /* Seconds -> HH-MM-SS */
      processedData->gpsEta->hour = toProcess[dGpsEta] / 3600;
      processedData->gpsEta->min = (toProcess[dGpsEta] / 60) - (processedData->gpsEta->hour * 60) ;
      processedData->gpsEta->sec = toProcess[dGpsEta] - 60 - (processedData->gpsEta->hour * 3600) - (processedData->gpsEta->min * 60) ;
      break;
    case dLatitude:
      /* Leaving identity for the moment */
      processedData->lat = toProcess[dLatitude];
      break;
    case dLongitude:
      /* Leaving identity for the moment */
      processedData->longi = toProcess[dLongitude];
      break;
    case dSimTime:
      /* Seconds -> HH-MM-SS */
      processedData->simulationTime->hour = toProcess[dSimTime] / 3600;
      processedData->simulationTime->min = (toProcess[dSimTime] / 60) - (processedData->simulationTime->hour * 60) ;
      processedData->simulationTime->sec = toProcess[dSimTime] - 60 - (processedData->simulationTime->hour * 3600) - (processedData->simulationTime->min * 60) ;
      break;
    case dTemperature:
      /* Celsius -> rounded to tenths Celsius */
      processedData->temp = toProcess[dTemperature];
      break;
    case dPressure:
      /* Millibars -> in Hg */
      processedData->pressure = toProcess[dPressure] / 33.864;
      break;
    case dWindVelocity:
      /* Feet per Second -> Knots  */
      processedData->windVelo = __double2int_rn(toProcess[dWindVelocity] / 1.688);
      break;
    case dWindDirection:
      /* Degrees -> rounded degrees */
      processedData->windDir = __double2int_rn(toProcess[dWindDirection]);
      break;
  }
}

void cudaProcess(simConnect::simData payload) {
  double *convertedData = toArray(payload);
  double *d_unprocessedData;
  processedData_t *d_processedData;
  absoluteTime_t *d_processedAbsolute;
  hTime_t *d_processedZulu;
  hTime_t *d_processedgpseta;
  hTime_t *d_processedsimulationTime;
  processedData_t  *processedData;
  int size = N * sizeof(double);

  /* Make space for our processed data */
  processedData = (processedData_t *) malloc(sizeof(processedData_t));
  processedData->absTime = (absoluteTime_t *) malloc(sizeof(absoluteTime_t));
  processedData->zulu = (hTime_t *) malloc(sizeof(hTime_t));
  processedData->gpsEta = (hTime_t *) malloc(sizeof(hTime_t));
  processedData->simulationTime = (hTime_t *) malloc(sizeof(hTime_t));
  /* Make space on GPU memory for our array */
  hipSuccess(hipMallocManaged((void **)&d_unprocessedData, size));
  /* Make space for processed data on the GPU */
  /* We use hipMallocManaged so we can derefference 
     the pointers on host or device                 */
  hipSuccess(hipMallocManaged((void **)&d_processedData, sizeof(processedData_t)));
  hipSuccess(hipMallocManaged((void **)&d_processedAbsolute, sizeof(absoluteTime_t)));
  hipSuccess(hipMallocManaged((void **)&d_processedZulu, sizeof(hTime_t)));
  hipSuccess(hipMallocManaged((void **)&d_processedgpseta, sizeof(hTime_t)));
  hipSuccess(hipMallocManaged((void **)&d_processedsimulationTime, sizeof(hTime_t)));
  /* Assign pointers*/
  d_processedData->absTime = d_processedAbsolute;
  d_processedData->zulu = d_processedZulu;
  d_processedData->gpsEta = d_processedgpseta;
  d_processedData->simulationTime = d_processedsimulationTime;
  /* Copy our data from host to device */
  hipSuccess(hipMemcpy(d_unprocessedData, convertedData, size, hipMemcpyHostToDevice));
  /* Launch kernel on the GPU to process the data we just copied */
  cudaHandle<<<N,1>>>(d_processedData ,d_unprocessedData);
  hipSuccess(hipGetLastError());
  /* Synchronize here so we're not touching managed memory */
  /* Make sure any device jobs finish before we cleanup and exit */
  hipSuccess(hipDeviceSynchronize());
  /* Log our processed object to file */
  logToFile(payload, convertedData, d_processedData); 
  /* Cleanup */
  free(processedData->absTime);
  free(processedData->zulu);
  free(processedData->gpsEta);
  free(processedData->simulationTime);
  free(processedData);
  free(convertedData);
  hipSuccess(hipFree(d_processedAbsolute));
  hipSuccess(hipFree(d_processedZulu));
  hipSuccess(hipFree(d_processedgpseta));
  hipSuccess(hipFree(d_processedsimulationTime));
  hipSuccess(hipFree(d_unprocessedData));
  hipSuccess(hipFree(d_processedData));
}

void cudaSchedule(int device, simConnect::simData payload) {
  hipSuccess(hipSetDevice(device));
  cudaProcess(payload);
}

int wrapperGetCudaDevCount() {
  int count = -1;
  hipSuccess(hipGetDeviceCount(&count));

  return count;
}