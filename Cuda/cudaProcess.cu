#include "hip/hip_runtime.h"
#include "cudaProcess.h"

void logToFile(simConnect::simData payload, double *convertedData, double *processedData ) {
  FILE *output = fopen("out.txt", "a+");
  std::string  printString ("");
  
  fprintf(output, "Object sztitle: %s\n", payload.sztitle().c_str());
  
  for (int i = 0; i < N; i++ ) {
    double toPrint;

    switch(i) {
      case dAbsoluteTime:
        toPrint = convertedData[i];
        printString = "Absolute time";
        break;
      case dTime:
        toPrint = convertedData[i];
        printString = "Zulu Time";
        break;
      case uSimOnGround:
        toPrint = convertedData[i];
        printString = "Sim On Ground";
        break;
      case dAltitude:
        toPrint = convertedData[i];
        printString = "Altitude";
        break;
      case dHeading:
        toPrint = convertedData[i];
        printString = "Heading";
        break;
      case dSpeed:
        toPrint = convertedData[i];
        printString = "Speed";
        break;
      case dVerticalSpeed:
        toPrint = convertedData[i];
        printString = "Vertical Speed";
        break;
      case dGpsEta:
        toPrint = convertedData[i];
        printString = "GPS ETA";
        break;
      case dLatitude:
        toPrint = convertedData[i];
        printString = "Latitude";
        break;
      case dLongitude:
        toPrint = convertedData[i];
        printString = "Longitude";
        break;
      case dSimTime:
        toPrint = convertedData[i];
        printString = "Sim time";
        break;
      case dTemperature:
        toPrint = convertedData[i];
        printString = "Temperature";
        break;
      case dPressure:
        toPrint = convertedData[i];
        printString = "Air Pressure";
        break;
      case dWindVelocity:
        toPrint = convertedData[i];
        printString = "Wind Velocity";
        break;
      case dWindDirection:
        toPrint = convertedData[i];
        printString = "Wind Direction";
        break;
    }

    fprintf(output, "Unprocessed %s %d: %f    ", printString.c_str(), i, toPrint);
    fprintf(output, "processedData %d: %f \n", i, processedData[i]);
  }

  fprintf(output, "\n\n");
  fclose(output);
}

double *toArray(simConnect::simData toConvert) {
  double *convertedData = (double *)malloc(sizeof(double) * N);

  for (int i = 0; i < N; i++ ) {
    switch(i) {
      case dAbsoluteTime:
        convertedData[i] = toConvert.dabsolutetime();
        break;
      case dTime:
        convertedData[i] = toConvert.dtime();
        break;
      case uSimOnGround:
        convertedData[i] = toConvert.usimonground();
        break;
      case dAltitude:
        convertedData[i] = toConvert.daltitude();
        break;
      case dHeading:
        convertedData[i] = toConvert.dheading();
        break;
      case dSpeed:
        convertedData[i] = toConvert.dspeed();
        break;
      case dVerticalSpeed:
        convertedData[i] = toConvert.dverticalspeed();
        break;
      case dGpsEta:
        convertedData[i] = toConvert.dgpseta();
        break;
      case dLatitude:
        convertedData[i] = toConvert.dlatitude();
        break;
      case dLongitude:
        convertedData[i] = toConvert.dlongitude();
        break;
      case dSimTime:
        convertedData[i] = toConvert.dsimtime();
        break;
      case dTemperature:
        convertedData[i] = toConvert.dtemperature();
        break;
      case dPressure:
        convertedData[i] = toConvert.dairpressure();
        break;
      case dWindVelocity:
        convertedData[i] = toConvert.dwindvelocity();
        break;
      case dWindDirection:
        convertedData[i] = toConvert.dwinddirection();
        break;
    }
  }

  return convertedData;
}

__global__ void cudaHandle(double * processedData, double *toProcess) {
  /* Process each field differently */
  switch(blockIdx.x) {
    case dAbsoluteTime:
      /* Seconds since 12:00AM Jan. 1 0000 -> YYYY-MM-DD-HH-MM-SS */
      
      processedData[dAbsoluteTime] = toProcess[dAbsoluteTime];
      break;
    case dTime:
      /* Seconds -> HH-MM-SS */
      processedData[dTime] = toProcess[dTime];
      break;
    case uSimOnGround:
      /* int -> bool */
      processedData[uSimOnGround] = toProcess[uSimOnGround];
      break;
    case dAltitude:
      /* feet -> rounded feet */
      processedData[dAltitude] = toProcess[dAltitude];
      break;
    case dHeading:
      /* Radians -> degrees */
      processedData[dHeading] = toProcess[dHeading];
      break;
    case dSpeed:
      /* Knots -> rounded knots */
      processedData[dSpeed] = toProcess[dSpeed];
      break;
    case dVerticalSpeed:
      /* Feet per Sec -> Feet Per Min */
      processedData[dVerticalSpeed] = toProcess[dVerticalSpeed];
      break;
    case dGpsEta:
      /* Seconds -> HH-MM-SS */
      processedData[dGpsEta] = toProcess[dGpsEta];
      break;
    case dLatitude:
      /* Leaving identity for the moment */
      processedData[dLatitude] = toProcess[dLatitude];
      break;
    case dLongitude:
      /* Leaving identity for the moment */
      processedData[dLongitude] = toProcess[dLongitude];
      break;
    case dSimTime:
      /* Seconds -> HH-MM-SS */
      processedData[dSimTime] = toProcess[dSimTime];
      break;
    case dTemperature:
      /* Celsius -> rounded to tenths Celsius */
      processedData[dTemperature] = toProcess[dTemperature];
      break;
    case dPressure:
      /* Millibars -> in Hg */
      processedData[dPressure] = toProcess[dPressure];
      break;
    case dWindVelocity:
      /* Feet per Second -> Knots  */
      processedData[dWindVelocity] = toProcess[dWindVelocity];
      break;
    case dWindDirection:
      /* Degrees -> rounded degrees */
      processedData[dWindDirection] = toProcess[dWindDirection];
      break;
  }
}

void cudaProcess(simConnect::simData payload) {
  double *convertedData = toArray(payload);
  double *d_unprocessedData;
  double *d_processedData;
  double *processedData;
  int size = N * sizeof(double);

  /* Make space for our processed data */
  processedData = (double *)malloc(size);
  /* Make space on GPU memory for our object */
  hipMalloc((void **)&d_unprocessedData, size);
  /* Make space for processed data on the GPU */
  hipMalloc((void **)&d_processedData, size);
  /* Copy our data from host to device */
  hipMemcpy(d_unprocessedData, convertedData, size, hipMemcpyHostToDevice);
  /* Launch kernel on the GPU to process the data we just copied */
  cudaHandle<<<N,1>>>(d_processedData ,d_unprocessedData);
  /* Copy our pocessed data back from device to host */
  hipMemcpy(processedData, d_processedData, size, hipMemcpyDeviceToHost);
  /* Make sure any device jobs finish before we cleanup and exit */
  hipDeviceSynchronize();
  /* Log our processed object to file */
  logToFile(payload, convertedData, processedData);

  /* Cleanup */
  free(processedData);
  free(convertedData);
  hipFree(d_processedData);
  hipFree(d_unprocessedData);
}
