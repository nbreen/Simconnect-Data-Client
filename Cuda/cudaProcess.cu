#include "hip/hip_runtime.h"
#include "cudaProcess.h"

__global__ void cudaHandle(double *toProcess) {
  toProcess[blockIdx.x] = toProcess[blockIdx.x];
}

double* toArray(simConnect::simData toConvert) {
  double *converted = (double *) malloc(sizeof(double) * N);
  converted[0] = toConvert.dabsolutetime();
  converted[1] = toConvert.dtime();
  converted[2] = toConvert.usimonground();
  converted[3] = toConvert.daltitude();
  converted[4] = toConvert.dheading();
  converted[5] = toConvert.dspeed();
  converted[6] = toConvert.dverticalspeed();
  converted[7] = toConvert.dgpseta();
  converted[8] = toConvert.dlatitude();
  converted[9] = toConvert.dlongitude();
  converted[10] = toConvert.dsimtime();
  converted[11] = toConvert.dtemperature();
  converted[12] = toConvert.dairpressure();
  converted[13] = toConvert.dwindvelocity();
  converted[14] = toConvert.dwinddirection();

  return converted;
}

void cudaProcess(simConnect::simData payload) {
  double *convertedData;
  double *d_convertedData;
  double *processedData;
  std::string  printString ("");
  int size = N * sizeof(double);
  FILE *output = fopen("out.txt", "a+");

  /* In order to process the data we first need it in an array for easier access */
  convertedData = toArray(payload);
  /* Make space for our processed data */
  processedData = (double *)malloc(size);
  /* Make space on GPU memory for our array */
  hipMalloc((void **)&d_convertedData, size);
  /* Copy our array from host to device */
  hipMemcpy(d_convertedData, convertedData, size, hipMemcpyHostToDevice);
  /* Launch kernel on the GPU to process the data we just copied */
  cudaHandle<<<N,1>>>(d_convertedData);
  /* Copy our pocessed data back from device to host */
  hipMemcpy(processedData, d_convertedData, size, hipMemcpyDeviceToHost);
  /* Make sure any device jobs finish before we cleanup and exit */
  hipDeviceSynchronize();

  fprintf(output, "Object sztitle: %s\n", payload.sztitle().c_str());
  for (int i = 0; i < N; i++ ) {
    double toPrint;

    switch(i) {
      case 0:
        toPrint = convertedData[i];
        printString = "Absolute time";
        break;
      case 1:
        toPrint = convertedData[i];
        printString = "Zulu Time";
        break;
      case 2:
        toPrint = convertedData[i];
        printString = "Sim On Ground";
        break;
      case 3:
        toPrint = convertedData[i];
        printString = "Altitude";
        break;
      case 4:
        toPrint = convertedData[i];
        printString = "Heading";
        break;
      case 5:
        toPrint = convertedData[i];
        printString = "Speed";
        break;
      case 6:
        toPrint = convertedData[i];
        printString = "Vertical Speed";
        break;
      case 7:
        toPrint = convertedData[i];
        printString = "GPS ETA";
        break;
      case 8:
        toPrint = convertedData[i];
        printString = "Latitude";
        break;
      case 9:
        toPrint = convertedData[i];
        printString = "Longitude";
        break;
      case 10:
        toPrint = convertedData[i];
        printString = "Sim time";
        break;
      case 11:
        toPrint = convertedData[i];
        printString = "Temperature";
        break;
      case 12:
        toPrint = convertedData[i];
        printString = "Air Pressure";
        break;
      case 13:
        toPrint = convertedData[i];
        printString = "Wind Velocity";
        break;
      case 14:
        toPrint = convertedData[i];
        printString = "Wind Direction";
        break;
    }
    fprintf(output, "Unprocessed %s %d: %f    ", printString.c_str(), i, toPrint);
    fprintf(output, "processedData %d: %f \n", i, processedData[i]);
  }

  /* Cleanup */
  fprintf(output, "\n\n");
  fclose(output);
  free(convertedData);
  free(processedData);
  hipFree(d_convertedData);

}
